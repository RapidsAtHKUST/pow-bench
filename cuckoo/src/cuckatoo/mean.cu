#include "hip/hip_runtime.h"
// Cuckatoo Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018-2019 Jiri Vadura (photon) and John Tromp
// This software is covered by the FAIR MINING license

#include <stdio.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include "cuckatoo.h"
#include "graph.hpp"
#include "../crypto/siphash.cuh"
#include "../crypto/blake2.h"

typedef uint8_t u8;
typedef uint16_t u16;

#ifndef NA
#define NA 4
#endif
#define NA2 (NA * NA)

#ifndef MAXSOLS
#define MAXSOLS 4
#endif

#ifndef IDXSHIFT
// number of bits of compression of surviving edge endpoints
// reduces space used in cycle finding, but too high a value
// results in NODE OVERFLOW warnings and fake cycles
#define IDXSHIFT 12
#endif

const u32 MAXEDGES = NEDGES >> IDXSHIFT;

typedef uint64_t u64; // save some typing

#ifndef XBITS
// assumes at least 2^18 bits of shared mem (32 KB) on thread block
// #define XBITS ((EDGEBITS-18+1)/2)
// scrap that; too few buckets inhibits parallellism
#define XBITS 6
#endif

const u32 NX        = 1 << XBITS;
const u32 XMASK     = NX - 1;
const u32 NX2       = NX * NX;
const u32 YBITS     = XBITS;
const u32 NY        = 1 << YBITS;
const u32 YZBITS    = EDGEBITS - XBITS;
const u32 ZBITS     = YZBITS - YBITS;
const u32 NZ        = 1 << ZBITS;
const u32 ZMASK     = NZ - 1;

#ifndef NEPS_A
#define NEPS_A 133
#endif
#ifndef NEPS_B
#define NEPS_B 85
#endif
#define NEPS 128

const u64 EDGES_A = NZ * NEPS_A / NEPS;
const u64 EDGES_B = NZ * NEPS_B / NEPS;

const u64 ROW_EDGES_A = EDGES_A * NY;
const u64 ROW_EDGES_B = EDGES_B * NY;

// Number of rows in bufferB not overlapping bufferA
#ifndef NRB1
#define NRB1 (NX / 2)
#endif
#define NRB2 (NX - NRB1)
#define NB 2

__constant__ uint2 recoveredges[PROOFSIZE];
__constant__ uint2 e0 = {0,0};

__device__ uint2 make_Edge(const u32 nonce, const uint2 dummy, const u32 node0, const u32 node1) {
   return make_uint2(node0, node1);
}

__device__ uint2 make_Edge(const uint2 edge, const uint2 dummy, const u32 node0, const u32 node1) {
   return edge;
}

__device__ u32 make_Edge(const u32 nonce, const u32 dummy, const u32 node0, const u32 node1) {
   return nonce;
}

#ifndef FLUSHA // should perhaps be in trimparams and passed as template parameter
#define FLUSHA 16
#endif

template<int maxOut, typename EdgeOut>
__global__ void SeedA(const siphash_keys &sipkeys, ulonglong4 * __restrict__ buffer, u32 * __restrict__ indexes) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int gid = group * dim + lid;
  const int nthreads = gridDim.x * dim;
  const int FLUSHA2 = 2*FLUSHA;

  __shared__ EdgeOut tmp[NX][FLUSHA2]; // needs to be ulonglong4 aligned
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(EdgeOut);
  __shared__ int counters[NX];

  for (int row = lid; row < NX; row += dim)
    counters[row] = 0;
  __syncthreads();

  const int col = group % NX;
  const int loops = NEDGES / nthreads; // assuming THREADS_HAVE_EDGES checked
  for (int i = 0; i < loops; i++) {
    u32 nonce = gid * loops + i;
    u32 node1, node0 = dipnode(sipkeys, (u64)nonce, 0);
    if (sizeof(EdgeOut) == sizeof(uint2)) {
      node1 = dipnode(sipkeys, (u64)nonce, 1);
    }
    int row = node0 >> YZBITS;
    int counter = min((int)atomicAdd(counters + row, 1), (int)(FLUSHA2-1)); // assuming ROWS_LIMIT_LOSSES checked
    tmp[row][counter] = make_Edge(nonce, tmp[0][0], node0, node1);
    __syncthreads();
    if (counter == FLUSHA-1) {
      int localIdx = min(FLUSHA2, counters[row]);
      int newCount = localIdx % FLUSHA;
      int nflush = localIdx - newCount;
      u32 grp = row * NX + col;
      int cnt = min((int)atomicAdd(indexes + grp, nflush), (int)(maxOut - nflush));
      for (int i = 0; i < nflush; i += TMPPERLL4) {
        buffer[((u64)grp * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
      }
      for (int t = 0; t < newCount; t++) {
        tmp[row][t] = tmp[row][t + nflush];
      }
      counters[row] = newCount;
    }
    __syncthreads();
  }
  EdgeOut zero = make_Edge(0, tmp[0][0], 0, 0);
  for (int row = lid; row < NX; row += dim) {
    int localIdx = min(FLUSHA2, counters[row]);
    u32 grp = row * NX + col;
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[row][j] = zero;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(indexes + grp, TMPPERLL4), (int)(maxOut - TMPPERLL4));
      buffer[((u64)grp * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
    }
  }
}

template <typename Edge> __device__ bool null(Edge e);

__device__ bool null(u32 nonce) {
  return nonce == 0;
}

__device__ bool null(uint2 nodes) {
  return nodes.x == 0 && nodes.y == 0;
}

#ifndef FLUSHB
#define FLUSHB 8
#endif

template<int maxOut, typename EdgeOut>
__global__ void SeedB(const siphash_keys &sipkeys, const EdgeOut * __restrict__ source, ulonglong4 * __restrict__ destination, const u32 * __restrict__ sourceIndexes, u32 * __restrict__ destinationIndexes) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int FLUSHB2 = 2 * FLUSHB;

  __shared__ EdgeOut tmp[NX][FLUSHB2];
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(EdgeOut);
  __shared__ int counters[NX];

  for (int col = lid; col < NX; col += dim)
    counters[col] = 0;
  __syncthreads();
  const int row = group / NX;
  const int bucketEdges = min((int)sourceIndexes[group], (int)maxOut);
  const int loops = (bucketEdges + dim-1) / dim;
  for (int loop = 0; loop < loops; loop++) {
    int col;
    int counter = 0;
    const int edgeIndex = loop * dim + lid;
    if (edgeIndex < bucketEdges) {
      const int index = group * maxOut + edgeIndex;
      EdgeOut edge = __ldg(&source[index]);
      if (!null(edge)) {
        u32 node0 = endpoint(sipkeys, edge, 0);
        col = (node0 >> ZBITS) & XMASK;
        counter = min((int)atomicAdd(counters + col, 1), (int)(FLUSHB2-1)); // assuming COLS_LIMIT_LOSSES checked
        tmp[col][counter] = edge;
      }
    }
    __syncthreads();
    if (counter == FLUSHB-1) {
      int localIdx = min(FLUSHB2, counters[col]);
      int newCount = localIdx % FLUSHB;
      int nflush = localIdx - newCount;
      u32 grp = row * NX + col;
      int cnt = min((int)atomicAdd(destinationIndexes + grp, nflush), (int)(maxOut - nflush));
      for (int i = 0; i < nflush; i += TMPPERLL4)
        destination[((u64)grp * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
      for (int t = 0; t < newCount; t++) {
        tmp[col][t] = tmp[col][t + nflush];
      }
      counters[col] = newCount;
    }
    __syncthreads(); 
  }
  EdgeOut zero = make_Edge(0, tmp[0][0], 0, 0);
  for (int col = lid; col < NX; col += dim) {
    int localIdx = min(FLUSHB2, counters[col]);
    u32 grp = row * NX + col;
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[col][j] = zero;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(destinationIndexes + grp, TMPPERLL4), (int)(maxOut - TMPPERLL4));
      destination[((u64)grp * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
    }
  }
}

__device__ __forceinline__  void bitmapset(u32 *ebitmap, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;
  atomicOr(ebitmap + word, mask);
}

__device__ __forceinline__  bool bitmaptest(u32 *ebitmap, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  return (ebitmap[word] >> bit) & 1;
}

template <typename Edge> u32 __device__ endpoint(const siphash_keys &sipkeys, Edge e, int uorv);

__device__ u32 endpoint(const siphash_keys &sipkeys, u32 nonce, int uorv) {
  return dipnode(sipkeys, nonce, uorv);
}

__device__ u32 endpoint(const siphash_keys &sipkeys, uint2 nodes, int uorv) {
  return uorv ? nodes.y : nodes.x;
}

__device__ u32 endpoint(uint2 nodes, int round) {
  return (round & 1) ? nodes.y : nodes.x;
}

#ifndef PART_BITS
// #bits used to partition edge set processing to save shared memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

const u32 PART_MASK = (1 << PART_BITS) - 1;
const u32 NONPART_BITS = ZBITS - PART_BITS;
const word_t NONPART_MASK = (1 << NONPART_BITS) - 1;
const int BITMAPBYTES = (NZ >> PART_BITS) / 8;

template<int maxIn, typename EdgeIn, int maxOut, typename EdgeOut>
__global__ void Round(const int round, const int part, const siphash_keys &sipkeys, EdgeIn * __restrict__ src, EdgeOut * __restrict__ dst, u32 * __restrict__ srcIdx, u32 * __restrict__ dstIdx) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int BITMAPWORDS = BITMAPBYTES / sizeof(u32);

  extern __shared__ u32 ebitmap[];

  for (int i = lid; i < BITMAPWORDS; i += dim)
    ebitmap[i] = 0;
  __syncthreads();
  int edgesInBucket = min(srcIdx[group], maxIn);
  // if (!group && !lid) printf("round %d size  %d\n", round, edgesInBucket);
  int loops = (edgesInBucket + dim-1) / dim;

  for (int loop = 0; loop < loops; loop++) {
    const int lindex = loop * dim + lid;
    if (lindex < edgesInBucket) {
      const int index = maxIn * group + lindex;
      EdgeIn edge = __ldg(&src[index]);
      if (null(edge)) continue;
      u32 z = endpoint(sipkeys, edge, round&1) & ZMASK;
      if ((z >> NONPART_BITS) == part) {
        bitmapset(ebitmap, z & NONPART_MASK);
      }
    }
  }
  __syncthreads();
  edgesInBucket = min(srcIdx[group], maxIn);
  loops = (edgesInBucket + dim-1) / dim;
  for (int loop = 0; loop < loops; loop++) {
    const int lindex = loop * dim + lid;
    if (lindex < edgesInBucket) {
      const int index = maxIn * group + lindex;
      EdgeIn edge = __ldg(&src[index]);
      if (null(edge)) continue;
      u32 node0 = endpoint(sipkeys, edge, round&1);
      u32 z = node0 & ZMASK;
      if ((z >> NONPART_BITS) == part && bitmaptest(ebitmap, (z & NONPART_MASK) ^ 1)) {
        u32 node1 = endpoint(sipkeys, edge, (round&1)^1);
        const int bucket = node1 >> ZBITS;
        const int bktIdx = min(atomicAdd(dstIdx + bucket, 1), maxOut - 1);
        dst[bucket * maxOut + bktIdx] = (round&1) ? make_Edge(edge, *dst, node1, node0) : make_Edge(edge, *dst, node0, node1);
      }
    }
  }
}

template<int maxIn0, int maxIn1, typename EdgeIn, int maxOut, typename EdgeOut>
__global__ void Round2(const int round, const int part, const siphash_keys &sipkeys, EdgeIn * __restrict__ src, EdgeOut * __restrict__ dst, u32 * __restrict__ srcIdx, u32 * __restrict__ dstIdx) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int BITMAPWORDS = BITMAPBYTES / sizeof(u32);

  extern __shared__ u32 ebitmap[];

  for (int i = lid; i < BITMAPWORDS; i += dim)
    ebitmap[i] = 0;
  __syncthreads();

  int edgesInBucket = min(srcIdx[group], maxIn0);
  // if (!group && !lid) printf("round %d size  %d\n", round, edgesInBucket);
  int loops = (edgesInBucket + dim-1) / dim;
  for (int loop = 0; loop < loops; loop++) {
    const int lindex = loop * dim + lid;
    if (lindex < edgesInBucket) {
      const int index = maxIn0 * group + lindex;
      EdgeIn edge = __ldg(&src[index]);
      if (null(edge)) continue;
      u32 z = endpoint(sipkeys, edge, round&1) & ZMASK;
      if ((z >> NONPART_BITS) == part) {
        bitmapset(ebitmap, z & NONPART_MASK);
      }
    }
  }
  edgesInBucket = min(srcIdx[NX2 + group], maxIn1);
  // if (!group && !lid) printf("round %d size  %d\n", round, edgesInBucket);
  loops = (edgesInBucket + dim-1) / dim;
  for (int loop = 0; loop < loops; loop++) {
    const int lindex = loop * dim + lid;
    if (lindex < edgesInBucket) {
      const int index = maxIn1 * group + lindex;
      EdgeIn edge = __ldg(&src[NX2*maxIn0 + index]);
      if (null(edge)) continue;
      u32 z = endpoint(sipkeys, edge, round&1) & ZMASK;
      if ((z >> NONPART_BITS) == part) {
        bitmapset(ebitmap, z & NONPART_MASK);
      }
    }
  }
  __syncthreads();

  edgesInBucket = min(srcIdx[group], maxIn0);
  loops = (edgesInBucket + dim-1) / dim;
  for (int loop = 0; loop < loops; loop++) {
    const int lindex = loop * dim + lid;
    if (lindex < edgesInBucket) {
      const int index = maxIn0 * group + lindex;
      EdgeIn edge = __ldg(&src[index]);
      if (null(edge)) continue;
      u32 node0 = endpoint(sipkeys, edge, round&1);
      u32 z = node0 & ZMASK;
      if ((z >> NONPART_BITS) == part && bitmaptest(ebitmap, (z & NONPART_MASK) ^ 1)) {
        u32 node1 = endpoint(sipkeys, edge, (round&1)^1);
        const int bucket = node1 >> ZBITS;
        const int bktIdx = min(atomicAdd(dstIdx + bucket, 1), maxOut - 1);
        dst[bucket * maxOut + bktIdx] = (round&1) ? make_Edge(edge, *dst, node1, node0) : make_Edge(edge, *dst, node0, node1);
      }
    }
  }
  edgesInBucket = min(srcIdx[NX2 + group], maxIn1);
  loops = (edgesInBucket + dim-1) / dim;
  for (int loop = 0; loop < loops; loop++) {
    const int lindex = loop * dim + lid;
    if (lindex < edgesInBucket) {
      const int index = maxIn1 * group + lindex;
      EdgeIn edge = __ldg(&src[NX2*maxIn0 + index]);
      if (null(edge)) continue;
      u32 node0 = endpoint(sipkeys, edge, round&1);
      u32 z = node0 & ZMASK;
      if ((z >> NONPART_BITS) == part && bitmaptest(ebitmap, (z & NONPART_MASK) ^ 1)) {
        u32 node1 = endpoint(sipkeys, edge, (round&1)^1);
        const int bucket = node1 >> ZBITS;
        const int bktIdx = min(atomicAdd(dstIdx + bucket, 1), maxOut - 1);
        dst[bucket * maxOut + bktIdx] = (round&1) ? make_Edge(edge, *dst, node1, node0) : make_Edge(edge, *dst, node0, node1);
      }
    }
  }
}

#ifndef LISTBITS
#define LISTBITS 12
#endif

const u32 NLISTS  = 1 << LISTBITS;
const u32 LISTMASK = NLISTS - 1;

#ifndef NNEXTS
#define NNEXTS NLISTS
#endif

template<int tpb, int bktInSize, int bktOutSize>
__global__  void Relay(const u32 round, const uint2 * source, uint2 * destination, const u32 * sourceIndexes, u32 * destinationIndexes, bool TAGGED)
{
  const int lid = threadIdx.x;
  const int group = blockIdx.x;

  __shared__ u32 lists[NLISTS];
  __shared__ u32 nexts[NNEXTS];

  const int nloops = (min(sourceIndexes[group], NNEXTS) - lid + tpb-1) / tpb;

  source += bktInSize * group;

  for (int i = 0; i < NLISTS/tpb; i++)
    lists[i * tpb + lid] = ~0;

  __syncthreads();

  for (int i = 0; i < nloops; i++) {
    const u32 index = i * tpb + lid;
    const u32 list = endpoint(source[index], round) & LISTMASK;
    nexts[index] = atomicExch(&lists[list], index);
  }

  __syncthreads();

  for (int i = nloops; --i >= 0;) {
    const u32 index = i * tpb + lid;
    const uint2 edge = source[index];
    const u32 edgex = endpoint(edge, round), edgey = endpoint(edge, round^1);
#ifdef DBG101
    if (((edgex^0x1d3cc2ae)&ZMASK)<2) printf("round %d group %x x %x y %x tag %x\n", round, group, edgex, edgey, edgex>>ZBITS);
    if (((edgey^0x1d3cc2ae)&ZMASK)<2) printf("round %d group %x x %x y %x tag %x\n", round, group, edgex, edgey, edgex>>ZBITS);
#endif
    if (edgey & NEDGES) continue; // copies don't relay
    u32 bucket = edgey >> ZBITS;
    u32 copybit = 0;
    const u32 list = (edgex & LISTMASK) ^ 1;
    for (u32 idx = lists[list]; idx != ~0; idx = nexts[idx]) {
      uint2 tagged = source[idx];
      const u32 taggedx = endpoint(tagged, round), taggedy = endpoint(tagged, round^1);
      if ((taggedx ^ edgex ^ 1) & ZMASK) continue;
      u32 bktIdx = min(atomicAdd(destinationIndexes + bucket, 1), bktOutSize - 1);
      u32 tag = TAGGED ? taggedx >> ZBITS : taggedy >> 1;
      destination[(bucket * bktOutSize) + bktIdx] = (round & 1)
	       ? make_uint2((tag << ZBITS) | (edge.x & ZMASK), copybit | (group << ZBITS) | (edge.y & ZMASK))
	       : make_uint2(copybit | (group << ZBITS) | (edge.x & ZMASK), (tag << ZBITS) | (edge.y & ZMASK));
      copybit = NEDGES;
    }
  }
}

template<int tpb, int maxIn>
__global__ void Tail(const uint2 *source, uint4 *destination, const u32 *sourceIndexes, u32 *destinationIndexes) {
  const int lid = threadIdx.x;
  const int group = blockIdx.x;

  __shared__ u32 lists[NLISTS];
  __shared__ u32 nexts[NNEXTS];

  const int nloops = (min(sourceIndexes[group], NNEXTS) - lid + tpb-1) / tpb;

  source += maxIn * group;

  for (int i = 0; i < NLISTS/tpb; i++)
    lists[i * tpb + lid] = ~0;

  __syncthreads();

  for (int i = 0; i < nloops; i++) {
    const u32 index = i * tpb + lid;
    const u32 list = source[index].x & LISTMASK;
    nexts[index] = atomicExch(&lists[list], index);
  }

  __syncthreads();

  for (int i = nloops; --i >= 0;) {
    const u32 index = i * tpb + lid;
    const uint2 edge = source[index];
#ifdef DBG101
    if (((edge.x^0x1d3cc2ae)&ZMASK)<2) printf("Tail group %x x %x y %x tag %x\n", group, edge.x, edge.y, edge.x>>ZBITS);
    if (((edge.y^0x1d3cc2ae)&ZMASK)<2) printf("Tail group %x x %x y %x tag %x\n", group, edge.x, edge.y, edge.x>>ZBITS);
#endif
    if (edge.x & 1) continue;
    const u32 list = (edge.x & LISTMASK) ^ 1;
    for (u32 idx = lists[list]; idx != ~0; idx = nexts[idx]) {
      uint2 other = source[idx];
      if ((other.x ^ edge.x) != 1) continue;
      u32 bktIdx = atomicAdd(destinationIndexes, 2);
      destination[bktIdx/2] = make_uint4(edge.y & (NEDGES-1), (group << ZBITS) |  (edge.x & ZMASK),
                                        other.y & (NEDGES-1), (group << ZBITS) | (other.x & ZMASK));
    }
  }
}

#define checkCudaErrors_V(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return;})
#define checkCudaErrors_N(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return NULL;})
#define checkCudaErrors(ans) ({int retval = gpuAssert((ans), __FILE__, __LINE__); if (retval != hipSuccess) return retval;})

inline int gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  int device_id;
  hipGetDevice(&device_id);
  if (code != hipSuccess) {
    snprintf(LAST_ERROR_REASON, MAX_NAME_LEN, "Device %d GPUassert: %s %s %d", device_id, hipGetErrorString(code), file, line);
    hipDeviceReset();
    if (abort) return code;
  }
  return code;
}

__global__ void Recovery(const siphash_keys &sipkeys, ulonglong4 *buffer, u32 *indexes) {
  const int gid = blockDim.x * blockIdx.x + threadIdx.x;
  const int lid = threadIdx.x;
  const int nthreads = blockDim.x * gridDim.x;
  const int loops = NEDGES / nthreads;
  __shared__ u32 nonces[PROOFSIZE];
  
  if (lid < PROOFSIZE) nonces[lid] = 0;
  __syncthreads();
  for (int i = 0; i < loops; i++) {
    u64 nonce = gid * loops + i;
    u64 u = dipnode(sipkeys, nonce, 0);
    u64 v = dipnode(sipkeys, nonce, 1);
    for (int i = 0; i < PROOFSIZE; i++) {
      if (recoveredges[i].x == v && recoveredges[i].y == u)
        nonces[i] = nonce;
    }
  }
  __syncthreads();
  if (lid < PROOFSIZE) {
    if (nonces[lid] > 0)
      indexes[lid] = nonces[lid];
  }
}

struct blockstpb {
  u16 blocks;
  u16 tpb;
};

struct trimparams {
  u16 ntrims;
  blockstpb genA;
  blockstpb genB;
  blockstpb trim;
  blockstpb tail;
  blockstpb recover;

  trimparams() {
    ntrims              =   31;
    genA.blocks         = 4096;
    genA.tpb            =  256;
    genB.blocks         =  NX2;
    genB.tpb            =  128;
    trim.blocks         =  NX2;
    trim.tpb            =  512;
    tail.blocks         =  NX2;
    tail.tpb            = 1024;
    recover.blocks      = 1024;
    recover.tpb         = 1024;
  }
};

typedef u32 proof[PROOFSIZE];

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  edgetrimmer *dt;
  size_t sizeA, sizeB;
  const size_t indexesSize = NX * NY * sizeof(u32);
  const size_t indexesSizeNA = NA * indexesSize;
  u8 *bufferA;
  u8 *bufferB;
  u8 *bufferAB;
  u32 *indexesA;
  u32 *indexesB;
  u32 nedges;
  u32 *uvnodes;
  siphash_keys sipkeys, *dipkeys;
  bool abort;
  bool initsuccess = false;

  edgetrimmer(const trimparams _tp) : tp(_tp) {
    checkCudaErrors_V(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors_V(hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(u32)));
    checkCudaErrors_V(hipMalloc((void**)&dipkeys, sizeof(siphash_keys)));
    checkCudaErrors_V(hipMalloc((void**)&indexesA, indexesSizeNA));
    checkCudaErrors_V(hipMalloc((void**)&indexesB, indexesSizeNA));
    sizeA = ROW_EDGES_A * NX * sizeof(u32);
    sizeB = ROW_EDGES_B * NX * sizeof(u32);
    const size_t nonoverlap = sizeB * NRB1 / NX;
    const size_t bufferSize = sizeA + nonoverlap;
    assert(bufferSize - sizeB >= sizeB / 2); // ensure enough space for Round 1, / 2 is for 0.296 / 0.632 without expansion
    checkCudaErrors_V(hipMalloc((void**)&bufferA, bufferSize));
    bufferAB = bufferA + nonoverlap;
    bufferB  = bufferA + bufferSize - sizeB;
    assert((NA & (NA-1)) == 0); // ensure NA is a 2 power
    assert(NA * NEPS_B * NRB1 >= NEPS_A * NX); // ensure disjoint source dest in SeedB
    assert(sizeA / NA <= nonoverlap); // equivalent to above
    assert(bufferA + sizeA * NRB2 / NX <= bufferB); // ensure disjoint source dest in 2nd phase of round 0
    assert(bufferA + sizeA == bufferB + sizeB * NRB2 / NX); // ensure alignment of overlap
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    initsuccess = true;
    int maxbytes = 0x10000; // 64 KB
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round<EDGES_A), uint2, EDGES_B*NRB1/NX, uint2>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round<EDGES_A),   u32, EDGES_B*NRB1/NX,   u32>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round<EDGES_A), uint2, EDGES_B*NRB2/NX, uint2>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round<EDGES_A),   u32, EDGES_B*NRB2/NX,   u32>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round2<EDGES_B*NRB2/NX), EDGES_B*NRB1/NX, uint2, EDGES_B/2, uint2>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round2<EDGES_B*NRB2/NX), EDGES_B*NRB1/NX,   u32, EDGES_B/2, uint2>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round2<EDGES_B*NRB2/NX), EDGES_B*NRB1/NX,   u32, EDGES_B/2,   u32>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round<EDGES_B/2), uint2, EDGES_A/4, uint2>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round<EDGES_B/2),   u32, EDGES_A/4, uint2>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round<EDGES_A/4), uint2, EDGES_B/4, uint2>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Round<EDGES_B/4), uint2, EDGES_B/4, uint2>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
  }
  u64 globalbytes() const {
    return (sizeA+sizeB*NRB1/NX) + 2 * indexesSizeNA + sizeof(siphash_keys) + PROOFSIZE * 2*sizeof(u32) + sizeof(edgetrimmer);
  }
  ~edgetrimmer() {
    checkCudaErrors_V(hipFree(bufferA));
    checkCudaErrors_V(hipFree(indexesA));
    checkCudaErrors_V(hipFree(indexesB));
    checkCudaErrors_V(hipFree(dipkeys));
    checkCudaErrors_V(hipFree(uvnodes));
    checkCudaErrors_V(hipFree(dt));
    hipDeviceReset();
  }


#ifndef RELAY_TPB
#define RELAY_TPB 512
#endif

#ifndef TAIL_TPB
#define TAIL_TPB 512
#endif

#ifndef VBIDX
#define VBIDX 0
#endif

  void indexcount(u32 round, const u32 *indexes) {
#ifdef VERBOSE
    u32 nedges;
    hipMemcpy(&nedges, indexes+VBIDX, sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    print_log("round %d edges %d\n", round, nedges);
#endif

  }
  u32 trim() {
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));
  
    hipMemcpy(dipkeys, &sipkeys, sizeof(sipkeys), hipMemcpyHostToDevice);
  
    hipDeviceSynchronize();
    float durationA, durationB;
    hipEventRecord(start, NULL);
  
    hipMemset(indexesB, 0, indexesSize);

    SeedA<EDGES_A,   u32><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, (ulonglong4*)bufferAB, (u32 *)indexesB);
  
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationA, start, stop);
    if (abort) return false;
    hipEventRecord(start, NULL);
  
    hipMemset(indexesA, 0, indexesSize);

    size_t qA = sizeA/NA;
    size_t qE = NX2 / NA;
    for (u32 i = 0; i < NA; i++) {
      SeedB<EDGES_A,   u32><<<tp.genB.blocks/NA, tp.genB.tpb>>>(*dipkeys, (const   u32 *)(bufferAB+i*qA), (ulonglong4*)(bufferA+i*qA), indexesB+i*qE, indexesA+i*qE);
      if (abort) return false;
    }
    indexcount(0, indexesA);

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationB, start, stop);
    checkCudaErrors(hipEventDestroy(start)); checkCudaErrors(hipEventDestroy(stop));
    print_log("Seeding completed in %.0f + %.0f ms\n", durationA, durationB);
    if (abort) return false;
  
    hipMemset(indexesB+NX2, 0, indexesSize);

    qA = sizeA * NRB2 / NX;
    qE = NX * NRB2;
    for (u32 part = 0; part <= PART_MASK; part++) {
      Round<EDGES_A,   u32, EDGES_B*NRB1/NX,   u32><<<tp.trim.blocks*NRB1/NX, tp.trim.tpb, BITMAPBYTES>>>(0, part, *dipkeys, (u32*)(bufferA+qA), (u32*)(bufferA+sizeA), indexesA+qE, indexesB+NX2); // to .632
      if (abort) return false;
    }
    indexcount(1, indexesB+NX2);

    hipMemset(indexesB, 0, indexesSize);

    for (u32 part = 0; part <= PART_MASK; part++) {
      Round<EDGES_A,   u32, EDGES_B*NRB2/NX,   u32><<<tp.trim.blocks*NRB2/NX, tp.trim.tpb, BITMAPBYTES>>>(0, part, *dipkeys, (u32*)bufferA, (u32*)bufferB, indexesA, indexesB); // to .632
      if (abort) return false;
    }
    indexcount(1, indexesB);

    hipMemset(indexesA, 0, indexesSize);

    for (u32 part = 0; part <= PART_MASK; part++) {
      Round2<EDGES_B*NRB2/NX, EDGES_B*NRB1/NX,   u32, EDGES_B/2,   u32><<<tp.trim.blocks, tp.trim.tpb, BITMAPBYTES>>>(1, part, *dipkeys, (  u32*)bufferB, (  u32*)bufferA, indexesB, indexesA); // to .296
      if (abort) return false;
    }
    indexcount(2, indexesA);

    hipMemset(indexesB, 0, indexesSize);

    for (u32 part = 0; part <= PART_MASK; part++) {
      Round<EDGES_B/2,   u32, EDGES_A/4, uint2><<<tp.trim.blocks, tp.trim.tpb, BITMAPBYTES>>>(2, part, *dipkeys, (u32 *)bufferA, (uint2 *)bufferB, indexesA, indexesB); // to .176
      if (abort) return false;
    }
    indexcount(3, indexesB);

    hipMemset(indexesA, 0, indexesSize);

    for (u32 part = 0; part <= PART_MASK; part++) {
      Round<EDGES_A/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb, BITMAPBYTES>>>(3, part, *dipkeys, (uint2 *)bufferB, (uint2 *)bufferA, indexesB, indexesA); // to .117
      if (abort) return false;
    }
    indexcount(4, indexesA);
  
    for (int round = 5; round < tp.ntrims + PROOFSIZE/2-1; round += 2) {
      hipMemset(indexesB, 0, indexesSize);
      if (round >= tp.ntrims)
        Relay<RELAY_TPB, EDGES_B/4, EDGES_B/4><<<NX2, RELAY_TPB>>>(round-1, (uint2 *)bufferA, (uint2 *)bufferB, indexesA, indexesB, round > tp.ntrims);
      else for (u32 part = 0; part <= PART_MASK; part++) {
        Round<EDGES_B/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb, BITMAPBYTES>>>(round-1, part, *dipkeys, (uint2 *)bufferA, (uint2 *)bufferB, indexesA, indexesB);
      }
      indexcount(round, indexesB);
      if (abort) return false;
      hipMemset(indexesA, 0, indexesSize);
      if (round+1 >= tp.ntrims)
        Relay<RELAY_TPB, EDGES_B/4, EDGES_B/4><<<NX2, RELAY_TPB>>>(round, (uint2 *)bufferB, (uint2 *)bufferA, indexesB, indexesA, round+1 > tp.ntrims);
      else for (u32 part = 0; part <= PART_MASK; part++) {
        Round<EDGES_B/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb, BITMAPBYTES>>>(round, part, *dipkeys, (uint2 *)bufferB, (uint2 *)bufferA, indexesB, indexesA);
      }
      indexcount(round+1, indexesA);
      if (abort) return false;
    }
    
    hipMemset(indexesB, 0, indexesSize);
    hipDeviceSynchronize();
  
    Tail<TAIL_TPB, EDGES_B/4><<<NX2, TAIL_TPB>>>((const uint2 *)bufferA, (uint4 *)bufferB, indexesA, indexesB);
    hipMemcpy(&nedges, indexesB, sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return nedges;
  }
};

struct solver_ctx {
  edgetrimmer trimmer;
  bool mutatenonce;
  uint2 *edges;
  graph<word_t> cg;
  uint2 soledges[PROOFSIZE];
  std::vector<u32> sols; // concatenation of all proof's indices

  solver_ctx(const trimparams tp, bool mutate_nonce) : trimmer(tp), cg(MAXEDGES, MAXEDGES, MAXSOLS, IDXSHIFT) {
    edges   = new uint2[MAXEDGES];
    mutatenonce = mutate_nonce;
  }

  void setheadernonce(char * const headernonce, const u32 len, const u32 nonce) {
    if (mutatenonce) {
      ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    }
    setheader(headernonce, len, &trimmer.sipkeys);
    sols.clear();
  }
  ~solver_ctx() {
    delete[] edges;
  }

  u32 findcycles(uint2 *edges, u32 nedges) {
    u32 ndupes = 0;
    cg.reset();
    for (u32 i = 0; i < nedges; i++)
      ndupes += !cg.add_compress_edge(edges[i].x, edges[i].y);
    for (u32 s = 0 ;s < cg.nsols; s++) {
#ifdef VERBOSE
      print_log("Solution");
#endif
      for (u32 j = 0; j < PROOFSIZE; j++) {
        soledges[j] = edges[cg.sols[s][j]];
#ifdef VERBOSE
	print_log(" (%x, %x)", soledges[j].x, soledges[j].y);
#endif
      }
#ifdef VERBOSE
      print_log("\n");
#endif
      sols.resize(sols.size() + PROOFSIZE);
      hipMemcpyToSymbol(HIP_SYMBOL(recoveredges), soledges, sizeof(soledges));
      hipMemset(trimmer.indexesB, 0, trimmer.indexesSize);
      Recovery<<<trimmer.tp.recover.blocks, trimmer.tp.recover.tpb>>>(*trimmer.dipkeys, (ulonglong4*)trimmer.bufferA, (u32 *)trimmer.indexesB);
      hipMemcpy(&sols[sols.size()-PROOFSIZE], trimmer.indexesB, PROOFSIZE * sizeof(u32), hipMemcpyDeviceToHost);
      checkCudaErrors(hipDeviceSynchronize());
      qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), cg.nonce_cmp);
    }
    return ndupes;
  }

  int solve() {
    u64 time0, time1;
    u32 timems,timems2;

    trimmer.abort = false;
    time0 = timestamp();
    u32 nedges = trimmer.trim();
    // if (!nedges)
      // return 0;
    if (nedges > MAXEDGES) {
      print_log("OOPS; losing %d edges beyond MAXEDGES=%d\n", nedges-MAXEDGES, MAXEDGES);
      nedges = MAXEDGES;
      return 0;
    }
    hipMemcpy(edges, trimmer.bufferB, nedges * 8, hipMemcpyDeviceToHost);
    time1 = timestamp(); timems  = (time1 - time0) / 1000000;
    time0 = timestamp();
    u32 ndupes = findcycles(edges, nedges);
    time1 = timestamp(); timems2 = (time1 - time0) / 1000000;
    print_log("%d trims %d ms %d edges %d dupes %d ms total %d ms\n", trimmer.tp.ntrims, timems, nedges, ndupes, timems2, timems+timems2);
    return sols.size() / PROOFSIZE;
  }

  void abort() {
    trimmer.abort = true;
  }

};

#include <unistd.h>

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

typedef solver_ctx SolverCtx;

CALL_CONVENTION int run_solver(SolverCtx* ctx,
                               char* header,
                               int header_length,
                               u32 nonce,
                               u32 range,
                               SolverSolutions *solutions,
                               SolverStats *stats
                               )
{
  u64 time0, time1;
  u32 timems;
  u32 sumnsols = 0;
  int device_id;
  if (stats != NULL) {
    hipGetDevice(&device_id);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, stats->device_id);
    stats->device_id = device_id;
    stats->edge_bits = EDGEBITS;
    strncpy(stats->device_name, props.name, MAX_NAME_LEN);
  }

  if (ctx == NULL || !ctx->trimmer.initsuccess){
    print_log("Error initialising trimmer. Aborting.\n");
    print_log("Reason: %s\n", LAST_ERROR_REASON);
    if (stats != NULL) {
       stats->has_errored = true;
       strncpy(stats->error_reason, LAST_ERROR_REASON, MAX_NAME_LEN);
    }
    return 0;
  }

  for (u32 r = 0; r < range; r++) {
    time0 = timestamp();
    ctx->setheadernonce(header, header_length, nonce + r);
    print_log("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r, ctx->trimmer.sipkeys.k0, ctx->trimmer.sipkeys.k1, ctx->trimmer.sipkeys.k2, ctx->trimmer.sipkeys.k3);
    u32 nsols = ctx->solve();
    time1 = timestamp();
    timems = (time1 - time0) / 1000000;
    print_log("Time: %d ms\n", timems);
    for (unsigned s = 0; s < nsols; s++) {
      print_log("Solution");
      u32* prf = &ctx->sols[s * PROOFSIZE];
      for (u32 i = 0; i < PROOFSIZE; i++)
        print_log(" %jx", (uintmax_t)prf[i]);
      print_log("\n");
      if (solutions != NULL){
        solutions->edge_bits = EDGEBITS;
        solutions->num_sols++;
        solutions->sols[sumnsols+s].nonce = nonce + r;
        for (u32 i = 0; i < PROOFSIZE; i++) 
          solutions->sols[sumnsols+s].proof[i] = (u64) prf[i];
      }
      int pow_rc = verify(prf, &ctx->trimmer.sipkeys);
      if (pow_rc == POW_OK) {
        print_log("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);
        for (int i=0; i<32; i++)
          print_log("%02x", cyclehash[i]);
        print_log("\n");
      } else {
        print_log("FAILED due to %s\n", errstr[pow_rc]);
      }
    }
    sumnsols += nsols;
    if (stats != NULL) {
      stats->last_start_time = time0;
      stats->last_end_time = time1;
      stats->last_solution_time = time1 - time0;
    }
  }
  print_log("%d total solutions\n", sumnsols);
  return sumnsols > 0;
}

CALL_CONVENTION SolverCtx* create_solver_ctx(SolverParams* params) {
  trimparams tp;
  tp.ntrims = params->ntrims;
  tp.genA.blocks = params->genablocks;
  tp.genA.tpb = params->genatpb;
  tp.genB.tpb = params->genbtpb;
  tp.trim.tpb = params->trimtpb;
  tp.tail.tpb = params->tailtpb;
  tp.recover.blocks = params->recoverblocks;
  tp.recover.tpb = params->recovertpb;

  hipDeviceProp_t prop;
  checkCudaErrors_N(hipGetDeviceProperties(&prop, params->device));

  assert(tp.genA.tpb <= prop.maxThreadsPerBlock);
  assert(tp.genB.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
  // assert(tp.tailblocks <= prop.threadDims[0]);
  assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
  assert(tp.recover.tpb <= prop.maxThreadsPerBlock);

  assert(tp.genA.blocks * tp.genA.tpb <= NEDGES); // check THREADS_HAVE_EDGES
  assert(tp.recover.blocks * tp.recover.tpb <= NEDGES); // check THREADS_HAVE_EDGES
  assert(tp.genA.tpb / NX <= FLUSHA); // check ROWS_LIMIT_LOSSES
  assert(tp.genA.tpb / NX <= FLUSHA); // check COLS_LIMIT_LOSSES

  hipSetDevice(params->device);
  if (!params->cpuload)
    checkCudaErrors_N(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

  SolverCtx* ctx = new SolverCtx(tp, params->mutate_nonce);

  return ctx;
}

CALL_CONVENTION void destroy_solver_ctx(SolverCtx* ctx) {
  delete ctx;
}

CALL_CONVENTION void stop_solver(SolverCtx* ctx) {
  ctx->abort();
}

CALL_CONVENTION void fill_default_params(SolverParams* params) {
  trimparams tp;
  params->device = 0;
  params->ntrims = tp.ntrims;
  params->genablocks = min(tp.genA.blocks, (u32)(NEDGES/tp.genA.tpb));
  params->genatpb = tp.genA.tpb;
  params->genbtpb = tp.genB.tpb;
  params->trimtpb = tp.trim.tpb;
  params->tailtpb = tp.tail.tpb;
  params->recoverblocks = min(tp.recover.blocks, (u32)(NEDGES/tp.recover.tpb));
  params->recovertpb = tp.recover.tpb;
  params->cpuload = false;
}

int main(int argc, char **argv) {
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len;
  int c;

  // set defaults
  SolverParams params;
  fill_default_params(&params);

  memset(header, 0, sizeof(header));
  while ((c = getopt(argc, argv, "scb:d:h:k:m:n:r:U:u:v:w:y:Z:z:")) != -1) {
    switch (c) {
      case 's':
        print_log("SYNOPSIS\n  cuda%d [-s] [-c] [-d device] [-h hexheader] [-m trims] [-n nonce] [-r range] [-U seedAblocks] [-u seedAthreads] [-v seedBthreads] [-w Trimthreads] [-y Tailthreads] [-Z recoverblocks] [-z recoverthreads]\n", EDGEBITS);
        print_log("DEFAULTS\n  cuda%d -d %d -h \"\" -m %d -n %d -r %d -U %d -u %d -v %d -w %d -y %d -Z %d -z %d\n", EDGEBITS, device, tp.ntrims, nonce, range, tp.genA.blocks, tp.genA.tpb, tp.genB.tpb, tp.trim.tpb, tp.tail.tpb, tp.recover.blocks, tp.recover.tpb);
        exit(0);
      case 'c':
        params.cpuload = false;
        break;
      case 'd':
        device = params.device = atoi(optarg);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        params.ntrims = atoi(optarg);
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'U':
        params.genablocks = atoi(optarg);
        break;
      case 'u':
        params.genatpb = atoi(optarg);
        break;
      case 'v':
        params.genbtpb = atoi(optarg);
        break;
      case 'w':
        params.trimtpb = atoi(optarg);
        break;
      case 'y':
        params.tailtpb = atoi(optarg);
        break;
      case 'Z':
        params.recoverblocks = atoi(optarg);
        break;
      case 'z':
        params.recovertpb = atoi(optarg);
        break;
    }
  }

  assert((params.ntrims & 1) == (PROOFSIZE/2 & 1)); // number of trims must match half cycle length in parity
  int nDevices;
  checkCudaErrors(hipGetDeviceCount(&nDevices));
  assert(device < nDevices);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 102400; dbytes>>=10,dunit++) ;
  print_log("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);

  print_log("Looking for %d-cycle on cuckatoo%d(\"%s\",%d", PROOFSIZE, EDGEBITS, header, nonce);
  if (range > 1)
    print_log("-%d", nonce+range-1);
  print_log(") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, params.ntrims, NX);

  SolverCtx* ctx = create_solver_ctx(&params);

  u64 bytes = ctx->trimmer.globalbytes();
  int unit;
  for (unit=0; bytes >= 102400; bytes>>=10,unit++) ;
  print_log("Using %d%cB of global memory.\n", (u32)bytes, " KMGT"[unit]);

  run_solver(ctx, header, sizeof(header), nonce, range, NULL, NULL);

  return 0;
}
